// Based upon following papers: http://worldcomp-proceedings.com/proc/p2011/CSC8087.pdf
// https://www.nvidia.com/content/PDF/isc-2011/Brandvik.pdf



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// I3D to index into a linear memory space from a 3D array index
#define I3D(i, j, k) ((i) + (N)*(j) + (N)*(N)*(k)) //newcode

#define N 16


double CLOCK() {
        struct timespec t;
        clock_gettime(CLOCK_MONOTONIC,  &t);
        return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}


// Following the HW structure, b_d is the data matrix and a_d will be the result matrix
__global__ void stencil(float *b, float *a) {
	int i, j, k, i000, im100, ip100, i0m10, i0p10, i00m1, i00p1;
	 // find i and j indices of this thread	 
	i = blockIdx.x * blockDim.x + threadIdx.x; 
	j = blockIdx.y * blockDim.y + threadIdx.y;
	k = blockIdx.z * blockDim.z + threadIdx.z;

	// find indices into linear memory for central point and neighbours
	
	 // i,j,k
	 i000 = I3D(i, j,k);
	 
	 // i-1, j, k
	 // i 'minus' 1, 0, 0
	 im100 = I3D(i-1, j,k);

	 // i+1, j,k
	 // i 'plus' 1, 0, 0 
	 ip100 = I3D(i+1, j,k);

	 // i, j-1, k
	 // 0, j 'minus' 1, k
	 i0m10 = I3D(i, j-1,k);

	 // i, j +1, k
	 i0p10 = I3D(i, j+1,k);
	  
	 // i, j, k-1
	 i00m1 = I3D(i, j,k-1);

	 // i, j, k+1
	 i00p1 = I3D(i, j,k+1);

	// checks all the variables aren't exceeding bounds (0 or outside tile)
	if (i > 0 && i < N-1 && j > 0 && j < N-1 && k> 0 && k < N-1) {
	
	// update temperatures
	a[i000] =  0.8*(b[im100] + b[ip100] 
					+ b[i0m10] + b[i0p10]
					+ b[i00m1] + b[i00p1]);
	}
}


int main(){

dim3 threadsPerBlock(N,N,N);

    double start, finish, total;
 
    float *h_a;
    float *h_b;
 
    float *d_a;
    float *d_b;
  
 
	// Size, in bytes, of each vector
    size_t bytes = N*sizeof(float);
 
	 // Allocate memory for each vector on host
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);

	 int i,j,k;
	 float a[N][N][N], b[N][N][N];

    for( i = 0; i < N; i++ ) {
        h_b[i] = 1;
    }

	for( i = 0; i < N; i++){
		for( j = 0; j < N; j++){
			for( k = 0 ; k < N; k++){
				b[i][j][k] = 1;
			}
		}
	}

	//CPU calculation to check for accuracy
for (i=1; i<N-1; i++)  
   for (j=1; j<N-1; j++)  
           for (k=1; k<N-1; k++) {  
 a[i][j][k]=0.8*(b[i-1][j][k]+b[i+1][j][k]+b[i][j-1][k] 
 + b[i][j+1][k]+b[i][j][k-1]+b[i][j][k+1]); 
     }  

	// Copy host vectors to device
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

	start = CLOCK();

	stencil<<<N, threadsPerBlock>>>(d_b, d_a);

	// Copy array back to host
    hipMemcpy( h_a, d_a, bytes, hipMemcpyDeviceToHost );

	finish = CLOCK();
	total = finish -start;
	printf("Time for the CUDA execution = %4.2f milliseconds\n", total);

	for (i=1; i<N-1; i++)  
   for (j=1; j<N-1; j++)  
     for (k=1; k<N-1; k++) {
			int ind = I3D(i,j,k);
			float op1 = a[i][j][k];
			float op2 = h_a[ind];
			if( op1 == op2){
			continue;
			}
			else{
				printf("Answer incorrect, %f != %f", a[i][j][k], h_a[ind]);
				return -1;
			}
     }  
	 printf("Wow all the numbers were the same, here's proof %f == %f", a[1][2][3], h_a[292]);

	// Release device memory
    hipFree(d_a);
    hipFree(d_b);
 
    // Release host memory
    free(h_a);
    free(h_b);
	free(a);
	free(b);
    return 0;
}


