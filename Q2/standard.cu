#include "hip/hip_runtime.h"
// Based upon following papers: http://worldcomp-proceedings.com/proc/p2011/CSC8087.pdf
// https://www.nvidia.com/content/PDF/isc-2011/Brandvik.pdf


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// I3D to index into a linear memory space from a 3D array index
#define I3D(i, j, k) ((i) + (N)*(j) + (N)*(N)*(k)) //newcode

#define N 16


double CLOCK() {
        struct timespec t;
        clock_gettime(CLOCK_MONOTONIC,  &t);
        return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}


// Following the HW structure, b_d is the data matrix and a_d will be the result matrix
__global__ void stencil(float *d_b, float *d_a) {
	int i, j, k, i000, im100, ip100, i0m10, i0p10, i00m1, i00p1;
	 // find i and j indices of this thread	 
	i = blockIdx.x * blockDim.x + threadIdx.x; 
	j = blockIdx.y * blockDim.y + threadIdx.y;
	k = blockIdx.z * blockDim.z + threadIdx.z;

	// find indices into linear memory for central point and neighbours
	
	 // i,j,k
	 i000 = I3D(i, j,k);
	 
	 // i-1, j, k
	 // i 'minus' 1, 0, 0
	 im100 = I3D(i-1, j,k);

	 // i+1, j,k
	 // i 'plus' 1, 0, 0 
	 ip100 = I3D(i+1, j,k);

	 // i, j-1, k
	 // 0, j 'minus' 1, k
	 i0m10 = I3D(i, j-1,k);

	 // i, j +1, k
	 i0p10 = I3D(i, j+1,k);
	  
	 // i, j, k-1
	 i00m1 = I3D(i, j,k-1);

	 // i, j, k+1
	 i00p1 = I3D(i, j,k+1);

	// checks all the variables aren't exceeding bounds (0 or outside tile)
	if (i > 0 && i < N-1 && j > 0 && j < N-1 && k> 0 && k < N-1) {
	
	// update temperatures
	d_a[i000] =  0.8*(d_b[im100] + d_b[ip100] 
					+ d_b[i0m10] + d_b[i0p10]
					+ d_b[i00m1] + d_b[i00p1]);
	}
}


int main(){

dim3 threadsPerBlock(N,N,N);

    double start, finish, total;
 
    float *h_a;
    float *h_b;
 
    float *d_a;
    float *d_b;
  
 
	// Size, in bytes, of each vector
    size_t bytes = N*sizeof(double);
 
	 // Allocate memory for each vector on host
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);

	 int i,j,k;
	 float a[N][N][N], b[N][N][N];

    for( i = 0; i < N; i++ ) {
        h_b[i] = 1;
    }

	for( i = 0; i < N; i++){
		for( j = 0; j < N; j++){
			for( k = 0 ; k < N; k++){
				b[i][j][k] = 1;
			}
		}
	}

	//CPU calculation to check for accuracy
for (i=1; i<N-1; i++)  
   for (j=1; j<N-1; j++)  
           for (k=1; k<n-1; k++) {  
 a[i][j][k]=0.8*(b[i-1][j][k]+b[i+1][j][k]+b[i][j-1][k] 
 + b[i][j+1][k]+b[i][j][k-1]+b[i][j][k+1]); 
     }  

	// Copy host vectors to device
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

	start = CLOCK();

	stencil<<<N, threadsPerBlock>>>(d_b, d_a);

	// Copy array back to host
    hipMemcpy( h_a, d_a, bytes, hipMemcpyDeviceToHost );

	finish = CLOCK();
	total = finish -start;
	printf("Time for the CUDA execution = %4.2f milliseconds\n", total);

	for (i=1; i<n-1; i++)  
   for (j=1; j<n-1; j++)  
     for (k=1; k<n-1; k++) {  
			if(a[i][j][k] == h_a[i][j][k]){
			continue;
			}
			else{
				printf("Answer incorrect");
				return -1;
			}
     }  


	// Release device memory
    hipFree(d_a);
    hipFree(d_b);
 
    // Release host memory
    free(h_a);
    free(h_b);
	free(a);
	free(b);
    return 0;
}


